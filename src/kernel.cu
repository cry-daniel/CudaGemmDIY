#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hipblas.h>
#include ""
#include <omp.h>
#include <iostream>

#define A100
#ifdef A100
#define SM_NUM   108
#define CUDA_CORE_PER_SM        64
#define CUDA_CORE_PER_WARP      16
#else
#define SM_NUM   40
#define CUDA_CORE_PER_SM        128
#define CUDA_CORE_PER_WARP      16
#endif

#define CHECK_CUDA(call) { \
    const hipError_t error = call; \
    if (error != hipSuccess) { \
        std::cerr << "Error: " << __FILE__ << ":" << __LINE__ << ", code: " << error \
                  << ", reason: " << hipGetErrorString(error) << std::endl; \
        exit(1); \
    } \
}

#define CHECK_CUBLAS(call) { \
    const hipblasStatus_t status = call; \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
        std::cerr << "Error: " << __FILE__ << ":" << __LINE__ << ", reason: " << status << std::endl; \
        exit(1); \
    } \
}

__global__ void blank_warmingGPU() {}

// A: row major; B: row major; C: row major;
void mulMatrixWithCpu(float* c, float* a, float* b, int m, int k, int n)
{
    int i_x = 0, i_y = 0;
#pragma omp parallel for private(i_x, i_y)
    for (int i = 0;i < m * n;i++)
    {
        i_x = i / n;  //i_x line of A,
        i_y = i % n;  //i_y column of B;
        for (int j = 0;j < k;j++)
        {
            c[i] += a[i_x * k + j] * b[j * n + i_y];
        }   
    }
}

// A: row major; B: row major; C: row major;
__global__ void mulKernel(float* c, float* a, float* b, int m, int k, int n)
{
    int i = 0;
    int j = 0;
    for(int index = blockIdx.x * blockDim.x + threadIdx.x;index < m * n;index+=gridDim.x*blockDim.x)
    {
        i = index / n;
        j = index % n;
        for (int l = 0; l < k; l++)
        {
            c[index] += a[i * k + l] * b[l * n + j];
        }
    }
}

// A: row major; B: row major; C: row major;
hipError_t mulMatrixWithNaiveCuda(float* c, float* a, float* b, int m, int k, int n)
{
    hipError_t cudaStatus = hipSuccess;
    blank_warmingGPU << <1, 1 >> > ();
    // create two events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // record start event on the default stream
    hipEventRecord(start);
    // execute kernel
    mulKernel << <SM_NUM, 32*CUDA_CORE_PER_SM / CUDA_CORE_PER_WARP >> > (c, a, b, m, k, n);
   // record stop event on the default stream
    hipEventRecord(stop);
    // wait until the stop event completes
    hipEventSynchronize(stop);
    // calculate the elapsed time between two events
    float time;
    hipEventElapsedTime(&time, start, stop);
    printf("Time_naive_cuda is %f ms.\n\n", time);
    // clean up the two events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());

    return cudaStatus;
}

// A: row major; B: row major; C: row major;
hipError_t mulMatrixWithCublasCuda(float* c, float* a, float* b, int m, int k, int n)
{
    hipError_t cudaStatus = hipSuccess;
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    float alpha = 1.0f;
    float beta = 0.0f;
    blank_warmingGPU << <1, 1 >> > ();
    // create two events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // record start event on the default stream
    hipEventRecord(start);
    // execute kernel
    // Result in row-major format
    CHECK_CUBLAS(hipblasGemmEx(handle,
                              HIPBLAS_OP_N, HIPBLAS_OP_N,
                              n, m, k,
                              &alpha,
                              b, HIP_R_32F, n,
                              a, HIP_R_32F, k,
                              &beta,
                              c, HIP_R_32F, n,
                              HIP_R_32F,
                              HIPBLAS_GEMM_DEFAULT));

    CHECK_CUBLAS(hipblasDestroy(handle));
   // record stop event on the default stream
    hipEventRecord(stop);
    // wait until the stop event completes
    hipEventSynchronize(stop);
    // calculate the elapsed time between two events
    float time;
    hipEventElapsedTime(&time, start, stop);
    printf("Time_cublas_cuda is %f ms.\n\n", time);
    // clean up the two events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());

    return cudaStatus;
}

// A: row major; B: row major; C: row major;
hipError_t mulMatrixWithCublasTC(float* c, float* a, float* b, int m, int k, int n)
{
    hipError_t cudaStatus = hipSuccess;
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    float alpha = 1.0f;
    float beta = 0.0f;
    blank_warmingGPU << <1, 1 >> > ();
    // create two events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // record start event on the default stream
    hipEventRecord(start);
    // execute kernel
    // Result in row-major format
    CHECK_CUBLAS(hipblasGemmEx(handle,
                              HIPBLAS_OP_N, HIPBLAS_OP_N,
                              n, m, k,
                              &alpha,
                              b, HIP_R_32F, n,
                              a, HIP_R_32F, k,
                              &beta,
                              c, HIP_R_32F, n,
                              HIP_R_32F,
                              CUBLAS_GEMM_DEFAULT_TENSOR_OP));

    CHECK_CUBLAS(hipblasDestroy(handle));
   // record stop event on the default stream
    hipEventRecord(stop);
    // wait until the stop event completes
    hipEventSynchronize(stop);
    // calculate the elapsed time between two events
    float time;
    hipEventElapsedTime(&time, start, stop);
    printf("Time_cublas_tc is %f ms.\n\n", time);
    // clean up the two events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());

    return cudaStatus;
}