#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hipblas.h>
#include ""
#include <omp.h>
#include <iostream>
#include <mma.h>

#define A100
#ifdef A100
#define SM_NUM   108
#define CUDA_CORE_PER_SM        64
#define CUDA_CORE_PER_WARP      16
#else
#define SM_NUM   40
#define CUDA_CORE_PER_SM        128
#define CUDA_CORE_PER_WARP      16
#endif

#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

using namespace nvcuda;

#define CHECK_CUDA(call) { \
    const hipError_t error = call; \
    if (error != hipSuccess) { \
        std::cerr << "Error: " << __FILE__ << ":" << __LINE__ << ", code: " << error \
                  << ", reason: " << hipGetErrorString(error) << std::endl; \
        exit(1); \
    } \
}

#define CHECK_CUBLAS(call) { \
    const hipblasStatus_t status = call; \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
        std::cerr << "Error: " << __FILE__ << ":" << __LINE__ << ", reason: " << status << std::endl; \
        exit(1); \
    } \
}

__global__ void blank_warmingGPU() {}

// A: row major; B: row major; C: row major;
void mulMatrixWithCpu(float* c, float* a, float* b, int m, int k, int n)
{
    int i_x = 0, i_y = 0;
#pragma omp parallel for private(i_x, i_y)
    for (int i = 0;i < m * n;i++)
    {
        i_x = i / n;  //i_x line of A,
        i_y = i % n;  //i_y column of B;
        for (int j = 0;j < k;j++)
        {
            c[i] += a[i_x * k + j] * b[j * n + i_y];
        }   
    }
}

// A: row major; B: row major; C: row major;
__global__ void mulKernel(float* c, float* a, float* b, int m, int k, int n)
{
    int i = 0;
    int j = 0;
    for(int index = blockIdx.x * blockDim.x + threadIdx.x;index < m * n;index+=gridDim.x*blockDim.x)
    {
        i = index / n;
        j = index % n;
        for (int l = 0; l < k; l++)
        {
            c[index] += a[i * k + l] * b[l * n + j];
        }
    }
}

// A: row major; B: row major; C: row major;
hipError_t mulMatrixWithNaiveCuda(float* c, float* a, float* b, int m, int k, int n)
{
    hipError_t cudaStatus = hipSuccess;
    blank_warmingGPU << <1, 1 >> > ();
    // create two events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // record start event on the default stream
    hipEventRecord(start);
    // execute kernel
    mulKernel << <SM_NUM, 32*CUDA_CORE_PER_SM / CUDA_CORE_PER_WARP >> > (c, a, b, m, k, n);
   // record stop event on the default stream
    hipEventRecord(stop);
    // wait until the stop event completes
    hipEventSynchronize(stop);
    // calculate the elapsed time between two events
    float time;
    hipEventElapsedTime(&time, start, stop);
    printf("Time_naive_cuda is %f ms.\n\n", time);
    // clean up the two events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());

    return cudaStatus;
}

// A: row major; B: row major; C: row major;
/* Cublas Cuda perform almost same as Cublas TC, since parameter 'hipblasGemmAlgo_t' doesn't have effect on NVIDIA Ampere architecture GPUs (A100) and newer. (https://docs.nvidia.com/cuda/cublas/index.html?highlight=gemmEx#cublasgemmalgo-t) */
hipError_t mulMatrixWithCublasCuda(float* c, float* a, float* b, int m, int k, int n)
{
    hipError_t cudaStatus = hipSuccess;
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    float alpha = 1.0f;
    float beta = 0.0f;
    blank_warmingGPU << <1, 1 >> > ();
    // create two events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // record start event on the default stream
    hipEventRecord(start);
    // execute kernel
    // Result in row-major format
    CHECK_CUBLAS(hipblasGemmEx(handle,
                              HIPBLAS_OP_N, HIPBLAS_OP_N,
                              n, m, k,
                              &alpha,
                              b, HIP_R_32F, n,
                              a, HIP_R_32F, k,
                              &beta,
                              c, HIP_R_32F, n,
                              HIP_R_32F,
                              HIPBLAS_GEMM_DEFAULT));

    CHECK_CUBLAS(hipblasDestroy(handle));
   // record stop event on the default stream
    hipEventRecord(stop);
    // wait until the stop event completes
    hipEventSynchronize(stop);
    // calculate the elapsed time between two events
    float time;
    hipEventElapsedTime(&time, start, stop);
    printf("Time_cublas_cuda is %f ms.\n\n", time);
    // clean up the two events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());

    return cudaStatus;
}

// A: row major; B: row major; C: row major;
hipError_t mulMatrixWithCublasTC(float* c, float* a, float* b, int m, int k, int n)
{
    hipError_t cudaStatus = hipSuccess;
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    float alpha = 1.0f;
    float beta = 0.0f;
    blank_warmingGPU << <1, 1 >> > ();
    // create two events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // record start event on the default stream
    hipEventRecord(start);
    // execute kernel
    // Result in row-major format
    CHECK_CUBLAS(hipblasGemmEx(handle,
                              HIPBLAS_OP_N, HIPBLAS_OP_N,
                              n, m, k,
                              &alpha,
                              b, HIP_R_32F, n,
                              a, HIP_R_32F, k,
                              &beta,
                              c, HIP_R_32F, n,
                              HIP_R_32F,
                              CUBLAS_GEMM_DEFAULT_TENSOR_OP));

    CHECK_CUBLAS(hipblasDestroy(handle));
   // record stop event on the default stream
    hipEventRecord(stop);
    // wait until the stop event completes
    hipEventSynchronize(stop);
    // calculate the elapsed time between two events
    float time;
    hipEventElapsedTime(&time, start, stop);
    printf("Time_cublas_tc is %f ms.\n\n", time);
    // clean up the two events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());

    return cudaStatus;
}

inline __device__ __host__ size_t div_ceil(size_t a, size_t b) {
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

__global__ void wmmaNaiveKernel(half *C, half *A, half *B, size_t M,
                                size_t K, size_t N) {
    const size_t K_tiles = div_ceil(K, WMMA_K);

    const size_t warp_row = blockIdx.y * WMMA_M;
    const size_t warp_col = blockIdx.x * WMMA_N;

    if (warp_row >= M && warp_col >= N) {
        return;
    }

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> C_frag;

    wmma::fill_fragment(C_frag, 0.0);

#pragma unroll
    for (size_t i = 0; i < K_tiles; ++i) {
        wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> A_frag;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> B_frag;

        wmma::load_matrix_sync(A_frag, A + warp_row * K + i * WMMA_K, K);
        wmma::load_matrix_sync(B_frag, B + i * WMMA_K + warp_col * K, K);

        wmma::mma_sync(C_frag, A_frag, B_frag, C_frag);
    }

    wmma::store_matrix_sync(C + warp_row * N + warp_col, C_frag, N, wmma::mem_row_major);
}

// A: row major; B: row major; C: row major;
hipError_t mulMatrixWithWmmaTC(half* c, half* a, half* b, size_t m, size_t k, size_t n)
{
    hipError_t cudaStatus = hipSuccess;
    blank_warmingGPU << <1, 1 >> > ();
    // create two events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // record start event on the default stream
    hipEventRecord(start);
    // execute kernel
    wmmaNaiveKernel << <SM_NUM, 32*CUDA_CORE_PER_SM / CUDA_CORE_PER_WARP >> > (c, a, b, m, k, n);
   // record stop event on the default stream
    hipEventRecord(stop);
    // wait until the stop event completes
    hipEventSynchronize(stop);
    // calculate the elapsed time between two events
    float time;
    hipEventElapsedTime(&time, start, stop);
    printf("Time_wmma_tc is %f ms.\n\n", time);
    // clean up the two events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());

    return cudaStatus;
}