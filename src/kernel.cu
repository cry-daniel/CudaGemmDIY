#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hipblas.h>
#include ""
#include <omp.h>
#include <iostream>
#include <mma.h>

#define A100
#ifdef A100
#define SM_NUM   108
#define CUDA_CORE_PER_SM        64
#define CUDA_CORE_PER_WARP      16
#else
#define SM_NUM   40
#define CUDA_CORE_PER_SM        128
#define CUDA_CORE_PER_WARP      16
#endif

#define WARP_SIZE 32
#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

using namespace nvcuda;

#define CHECK_CUDA(call) { \
    const hipError_t error = call; \
    if (error != hipSuccess) { \
        std::cerr << "Error: " << __FILE__ << ":" << __LINE__ << ", code: " << error \
                  << ", reason: " << hipGetErrorString(error) << std::endl; \
        exit(1); \
    } \
}

#define CHECK_CUBLAS(call) { \
    const hipblasStatus_t status = call; \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
        std::cerr << "Error: " << __FILE__ << ":" << __LINE__ << ", reason: " << status << std::endl; \
        exit(1); \
    } \
}

__global__ void blank_warmingGPU() {}

// A: row major; B: col major; C: row major;
template <typename T>
void mulMatrixWithCpu(T* c, T* a, T* b, int m, int k, int n)
{
    int i_x = 0, i_y = 0;
#pragma omp parallel for private(i_x, i_y)
    for (int i = 0;i < m * n;i++)
    {
        i_x = i / n;  //i_x line of A,
        i_y = i % n;  //i_y column of B;
        for (int j = 0;j < k;j++)
        {
            c[i] += a[i_x * k + j] * b[j + i_y * n];
        }   
    }
}

// A: row major; B: col major; C: row major;
template <typename T>
__global__ void mulKernel(T* c, T* a, T* b, int m, int k, int n)
{
    int i = 0;
    int j = 0;
    for(int index = blockIdx.x * blockDim.x + threadIdx.x;index < m * n;index+=gridDim.x*blockDim.x)
    {
        i = index / n;
        j = index % n;
        for (int l = 0; l < k; l++)
        {
            c[index] += a[i * k + l] * b[l + j * n];
        }
    }
}

// A: row major; B: col major; C: row major;
template <typename T>
hipError_t mulMatrixWithNaiveCuda(T* c, T* a, T* b, size_t m, size_t k, size_t n)
{
    hipError_t cudaStatus = hipSuccess;
    blank_warmingGPU << <1, 1 >> > ();
    // create two events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // record start event on the default stream
    hipEventRecord(start);
    // execute kernel
    mulKernel << <SM_NUM, 32*CUDA_CORE_PER_SM / CUDA_CORE_PER_WARP >> > (c, a, b, m, k, n);
   // record stop event on the default stream
    hipEventRecord(stop);
    // wait until the stop event completes
    hipEventSynchronize(stop);
    // calculate the elapsed time between two events
    float time;
    hipEventElapsedTime(&time, start, stop);
    printf("Time_naive_cuda is %f ms.\n\n", time);
    // clean up the two events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());

    return cudaStatus;
}

// A: row major; B: col major; C: row major;
/* Cublas Cuda perform almost same as Cublas TC, since parameter 'hipblasGemmAlgo_t' doesn't have effect on NVIDIA Ampere architecture GPUs (A100) and newer. (https://docs.nvidia.com/cuda/cublas/index.html?highlight=gemmEx#cublasgemmalgo-t) */
hipError_t mulMatrixWithCublasCuda(float* c, float* a, float* b, int m, int k, int n)
{
    hipError_t cudaStatus = hipSuccess;
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    float alpha = 1.0f;
    float beta = 0.0f;
    blank_warmingGPU << <1, 1 >> > ();
    // create two events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // record start event on the default stream
    hipEventRecord(start);
    // execute kernel
    // Result in row-major format
    CHECK_CUBLAS(hipblasGemmEx(handle,
                              HIPBLAS_OP_T, HIPBLAS_OP_N,
                              n, m, k,
                              &alpha,
                              b, HIP_R_32F, n,
                              a, HIP_R_32F, k,
                              &beta,
                              c, HIP_R_32F, n,
                              HIP_R_32F,
                              HIPBLAS_GEMM_DEFAULT));

    CHECK_CUBLAS(hipblasDestroy(handle));
   // record stop event on the default stream
    hipEventRecord(stop);
    // wait until the stop event completes
    hipEventSynchronize(stop);
    // calculate the elapsed time between two events
    float time;
    hipEventElapsedTime(&time, start, stop);
    printf("Time_cublas_cuda is %f ms.\n\n", time);
    // clean up the two events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());

    return cudaStatus;
}

// A: row major; B: col major; C: row major;
hipError_t mulMatrixWithCublasTC(float* c, float* a, float* b, int m, int k, int n)
{
    hipError_t cudaStatus = hipSuccess;
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    float alpha = 1.0f;
    float beta = 0.0f;
    blank_warmingGPU << <1, 1 >> > ();
    // create two events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // record start event on the default stream
    hipEventRecord(start);
    // execute kernel
    // Result in row-major format
    CHECK_CUBLAS(hipblasGemmEx(handle,
                              HIPBLAS_OP_T, HIPBLAS_OP_N,
                              n, m, k,
                              &alpha,
                              b, HIP_R_32F, n,
                              a, HIP_R_32F, k,
                              &beta,
                              c, HIP_R_32F, n,
                              HIP_R_32F,
                              CUBLAS_GEMM_DEFAULT_TENSOR_OP));

    CHECK_CUBLAS(hipblasDestroy(handle));
   // record stop event on the default stream
    hipEventRecord(stop);
    // wait until the stop event completes
    hipEventSynchronize(stop);
    // calculate the elapsed time between two events
    float time;
    hipEventElapsedTime(&time, start, stop);
    printf("warm up\n");
    // printf("Time_cublas_tc is %f ms.\n\n", time);
    // clean up the two events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());

    return cudaStatus;
}

inline __device__ __host__ size_t div_ceil(size_t a, size_t b) {
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

template <typename T>
__global__ void wmmaNaiveKernel(T *C, T *A, T *B, size_t M,
                                size_t K, size_t N) {
    for (int i_ = 0; i_ < 60; i_++){
        const size_t K_tiles = div_ceil(K, WMMA_K);

        const size_t warp_row = blockIdx.y * WMMA_M;
        const size_t warp_col = blockIdx.x * WMMA_N;

        if (warp_row >= M && warp_col >= N) {
            return;
        }

        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, T> C_frag;

        wmma::fill_fragment(C_frag, 0.0);

#pragma unroll
        for (size_t i = 0; i < K_tiles; ++i) {
            wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, T, wmma::row_major> A_frag;
            wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, T, wmma::col_major> B_frag;

            wmma::load_matrix_sync(A_frag, A + warp_row * K + i * WMMA_K, K);
            wmma::load_matrix_sync(B_frag, B + i * WMMA_K + warp_col * K, K);

            wmma::mma_sync(C_frag, A_frag, B_frag, C_frag);
        }

        wmma::store_matrix_sync(C + warp_row * N + warp_col, C_frag, N, wmma::mem_row_major);
    }
}

// A: row major; B: col major; C: row major;
template <typename T>
hipError_t mulMatrixWithWmmaTC(T* c, T* a, T* b, size_t m, size_t k, size_t n)
{
    hipError_t cudaStatus = hipSuccess;
    blank_warmingGPU << <1, 1 >> > ();
    // create two events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // record start event on the default stream
    hipEventRecord(start);
    // execute kernel
    // wmmaNaiveKernel << <SM_NUM, 32*CUDA_CORE_PER_SM / CUDA_CORE_PER_WARP >> > (c, a, b, m, k, n);
    dim3 block(WARP_SIZE);
    dim3 grid(div_ceil(n, WMMA_N), div_ceil(m, WMMA_M));
    wmmaNaiveKernel<<<grid, block>>>(c, a, b, m, k, n);
    // record stop event on the default stream
    hipEventRecord(stop);
    // wait until the stop event completes
    hipEventSynchronize(stop);
    // calculate the elapsed time between two events
    float time;
    hipEventElapsedTime(&time, start, stop);
    printf("Time_wmma_tc is %f ms.\n\n", time);
    // clean up the two events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());

    return cudaStatus;
}

template <typename T>
hipError_t mulMatrixWithSeqWmmaTC(T* c, T* c_, T* a, T* b, size_t m, size_t k, size_t n)
{
    hipError_t cudaStatus = hipSuccess;
    // blank_warmingGPU << <1, 1 >> > ();
    // create two events
    hipEvent_t start, stop, stop2;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&stop2);
    // record start event on the default stream
    hipEventRecord(start);
    // execute kernel
    // wmmaNaiveKernel << <SM_NUM, 32*CUDA_CORE_PER_SM / CUDA_CORE_PER_WARP >> > (c, a, b, m, k, n);
    dim3 block(WARP_SIZE);
    dim3 grid(div_ceil(n, WMMA_N), div_ceil(m, WMMA_M));
    mulKernel<<<SM_NUM, 32*CUDA_CORE_PER_SM / CUDA_CORE_PER_WARP>>>(c, a, b, m, k, n);
    hipEventRecord(stop2);
    wmmaNaiveKernel<<<grid, block>>>(c_, a, b, m, k, n);
    // record stop event on the default stream
    hipEventRecord(stop);
    // wait until the stop event completes
    hipEventSynchronize(stop);
    // calculate the elapsed time between two events
    float time, time2;
    hipEventElapsedTime(&time, start, stop);
    hipEventElapsedTime(&time2, start, stop2);
    printf("Time_seq_wmma_tc is %f ms & %f ms.\n\n", time2, time);
    // clean up the two events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(stop2);

    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());

    return cudaStatus;
}

template <typename T>
hipError_t mulMatrixWithMixedWmmaTC(T* c, T* c_, T* a, T* b, size_t m, size_t k, size_t n)
{
    hipStream_t streams[2];
    for (int i = 0; i < 2; i++) {
        CHECK_CUDA(hipStreamCreate(&streams[i]));
    }
    hipError_t cudaStatus = hipSuccess;
    // blank_warmingGPU << <1, 1 >> > ();
    // create two events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // record start event on the default stream
    hipEventRecord(start);
    // execute kernel
    // wmmaNaiveKernel << <SM_NUM, 32*CUDA_CORE_PER_SM / CUDA_CORE_PER_WARP >> > (c, a, b, m, k, n);
    dim3 block(WARP_SIZE);
    dim3 grid(div_ceil(n, WMMA_N), div_ceil(m, WMMA_M));
    mulKernel<<<SM_NUM, 32*CUDA_CORE_PER_SM / CUDA_CORE_PER_WARP, 0, streams[0]>>>(c, a, b, m, k, n);
    wmmaNaiveKernel<<<grid, block, 0, streams[1]>>>(c_, a, b, m, k, n);
    // record stop event on the default stream
    hipEventRecord(stop);
    // wait until the stop event completes
    hipEventSynchronize(stop);
    // calculate the elapsed time between two events
    float time;
    hipEventElapsedTime(&time, start, stop);
    printf("Time_mixed_wmma_tc is %f ms.\n\n", time);
    // clean up the two events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());

    return cudaStatus;
}